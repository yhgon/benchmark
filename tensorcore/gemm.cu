// build with command  nvcc -lcublas -lcudart -lcurand -arch=sm_70 gemm.cu
// use max clock 

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>

// Must be multiples of 16 to fit TensorCore
#define SIZE 8192  //  4096 8192 10240 16384 24576
#define MATRIX_M SIZE 
#define MATRIX_N SIZE
#define MATRIX_K SIZE

#define num_clock 1530 // V100 16GB SMX
#define num_SM 80
#define num_CUDA 64
#define num_FMA 2
#define num_DP_ratio 2
#define FP32_OP  num_clock*num_SM*num_CUDA*num_FMA
#define FP64_OP  FP32_OP / num_DP_ratio
#define TOTAL_OP  MATRIX_M * MATRIX_N * MATRIX_K * 2
#define TOTAL_OP2 (MATRIX_M*MATRIX_N) * (2*MATRIX_K+2) 

int main(int argc, char* argv[]) {

   printf("FP64 Matrix Memory Size : %.1f \n",  (float) (sizeof(double) * (float) (MATRIX_M*MATRIX_M)  / ( 1024 * 1024 ) )  );
   printf("FP32 Matrix Memory Size : %.1f \n",  (float) (sizeof(float) * (float) (MATRIX_M*MATRIX_M)  / ( 1024 * 1024 ) )  );
	
   double *a_fp64;
   double *b_fp64;
   double *c_fp64;
   float *a_fp32;
   float *b_fp32;
   float *c_fp32;
   double *c_cublas_fp64;
   float *c_cublas_fp32;
   double *c_host_cublas_fp64; // for error tolorence 
   float *c_host_cublas_fp32; // for error tolorence 

   printf(" Step1. Initialize GPU API handles...\n");
   hiprandGenerator_t gen;

   hipblasHandle_t cublasHandle;
   hipblasCreate(&cublasHandle);
       
   hipEvent_t startcublas_fp64;
   hipEvent_t startcublas_fp32;
   hipEvent_t stopcublas_fp64;   
   hipEvent_t stopcublas_fp32;      

   hipEventCreate(&startcublas_fp64);
   hipEventCreate(&startcublas_fp32);
   hipEventCreate(&stopcublas_fp64);
   hipEventCreate(&stopcublas_fp32);
   
   // Use tensor cores
   // cublasSetMathMode(cublasHandle, CUBLAS_TENSOR_OP_MATH);

   printf(" Step2. Memory Mallocation ...\n");
   hipMalloc((void**)&a_fp64, MATRIX_M * MATRIX_K * sizeof(double));
   hipMalloc((void**)&b_fp64, MATRIX_K * MATRIX_N * sizeof(double));
   hipMalloc((void**)&c_fp64, MATRIX_M * MATRIX_N * sizeof(double));
	 
   hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float));
   hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float));
   hipMalloc((void**)&c_fp32, MATRIX_M * MATRIX_N * sizeof(float));
	 
   hipMalloc((void**)&c_cublas_fp64, MATRIX_M * MATRIX_N * sizeof(double));
   hipMalloc((void**)&c_cublas_fp32, MATRIX_M * MATRIX_N * sizeof(float));

   c_host_cublas_fp64 = (double*)malloc(MATRIX_M * MATRIX_N * sizeof(double));
   c_host_cublas_fp32 = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

   printf(" Step3. Data init with cuRAND ...\n");
   hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
   hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL);

   hiprandGenerateUniformDouble(gen, a_fp64, MATRIX_M * MATRIX_K);
   hiprandGenerateUniformDouble(gen, b_fp64, MATRIX_K * MATRIX_N);
   hiprandGenerateUniformDouble(gen, c_fp64, MATRIX_M * MATRIX_N);

   hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K);
   hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N);
   hiprandGenerateUniform(gen, c_fp32, MATRIX_M * MATRIX_N);

   hipMemcpy(c_cublas_fp64, c_fp64, MATRIX_M * MATRIX_N * sizeof(double), hipMemcpyDeviceToDevice);
   hipMemcpy(c_cublas_fp32, c_fp32, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice);
	 
   hiprandDestroyGenerator(gen);

   double alpha_fp64 = 2.0f;
   double  beta_fp64 = 2.0f;

   float alpha_fp32 = 2.0f;
   float  beta_fp32 = 2.0f;

   printf(" Step5. Ready to Run...\n");
   printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha_fp64, beta_fp64);

   // Now using cuBLAS
   printf("warm up...");
   hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
               MATRIX_M, MATRIX_N, MATRIX_K, 
  	       &alpha_fp32, 
	       a_fp32, MATRIX_M, 
	       b_fp32, MATRIX_K, 
	       &beta_fp32, 
	       c_fp32, MATRIX_M);
								
    printf(" Step6.  Running with cuBLAS... sgemm\n");
    hipEventRecord(startcublas_fp32);
    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
	        &alpha_fp32, 
		a_fp32, MATRIX_M, 
		b_fp32, MATRIX_K, 
		&beta_fp32, 
		c_fp32, MATRIX_M);
   hipEventRecord(stopcublas_fp32);							
   hipEventSynchronize(stopcublas_fp32);

	 
   printf(" Step6.  Running with cuBLAS...dgemm\n");
   hipEventRecord(startcublas_fp64);
   hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
               MATRIX_M, MATRIX_N, MATRIX_K, 
	       &alpha_fp64, 
	       a_fp64, MATRIX_M, 
	       b_fp64, MATRIX_K, 
	       &beta_fp64, 
	       c_cublas_fp64, MATRIX_M);
   hipEventRecord(stopcublas_fp64);
   hipEventSynchronize(stopcublas_fp64);
	 
   printf(" Step7. Download results...\n");
   hipMemcpy( c_host_cublas_fp64, c_cublas_fp64, MATRIX_M * MATRIX_N * sizeof(double), hipMemcpyDeviceToHost);
   hipMemcpy( c_host_cublas_fp32, c_cublas_fp32, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost);
	 
   float cublasTime_fp32, cublasTime_fp64;
   hipEventElapsedTime(&cublasTime_fp32, startcublas_fp32, stopcublas_fp32);
   hipEventElapsedTime(&cublasTime_fp64, startcublas_fp64, stopcublas_fp64);

   printf("SGEMM cublas took %fms\n", cublasTime_fp32);
   printf("DGEMM cublas took %fms  ", cublasTime_fp64);
   printf(" with   %f Operation and clock %d Mhz \n", (double) TOTAL_OP,  num_clock );

   printf("FP64 RPeak: %.2f TFLOPS",  (double) FP64_OP /(1000000)  );
   printf(" DGEMM : %.2f TFLOPS\n",   (double) ( ((double)TOTAL_OP / (double) (1000000) ) / ((double) cublasTime_fp64)/1000 ) ); 
   printf("FP32 RPeak: %.2f TFLOPS",  (double) FP32_OP /(1000000)   );  
   printf(" SGEMM : %.2f TFLOPS\n",   (double) ( ((double)TOTAL_OP / (double) (1000000) ) / ((double) cublasTime_fp32)/1000 ) ); 

   hipEventDestroy(startcublas_fp64);
   hipEventDestroy(startcublas_fp32);
   hipEventDestroy(stopcublas_fp64);             
   hipEventDestroy(stopcublas_fp32);
   
   hipFree(a_fp64);
   hipFree(b_fp64);
   hipFree(c_fp64);
	 
   hipFree(a_fp32);
   hipFree(b_fp32);
   hipFree(c_fp32);
	 
   hipFree(c_cublas_fp64);
   hipFree(c_cublas_fp32);
   free(c_host_cublas_fp64);
   free(c_host_cublas_fp32);

   hipDeviceReset();
   return 0;
}
