/*
compile : nvcc malloc_test.cu -o malloc_test
exec : ./malloc_test 512 1
starting 512MB , increase 1MB 
if you have multiple GPU, use below
export CUDA_VISIBLE_DEVICES=3;./malloc_test 512 1

MB convert table would be : 
GB	MB
1	1024
2	2048
3	3072
4	4096
5	5120
6	6144
7	7168
8	8192
9	9216
10	10240
11	11264
12	12288
13	13312
14	14336
15	15360
16	16384
17	17408
18	18432
19	19456
20	20480
21	21504
22	22528
23	23552
24	24576
25	25600
26	26624
27	27648
28	28672
29	29696
30	30720
31	31744
32	32768
*/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

int main(int argc, char* argv[]) {

	int *devPtr= NULL;
	size_t mem_size, free, total , start, inc, size ;
	float free_f, total_f, used_f, used_start_f, used_now_f;

        hipMemGetInfo(&free,&total);
        free_f  = float(free)   / (1024*1024) ; 
        total_f = float(total)  / (1024*1024) ;
        used_f =  total_f-free_f ;
	used_start_f  = used_f;

	if(argc<3){
	printf(" plz use below command \n ./malloc_test 1024 10  \n to  malloc 1024MB and increment would be 10MB \n");
	printf("\n11GB 11264MB\n15GB 15360MB \n23GB 23552MB\n31GB 31744MB\n");
	printf("\ncurrent free memory is %.1f\n", free_f);

	return 0;
	}

        start =  atoi(argv[1]) / sizeof(int) ;
        inc = atoi(argv[2])  ;
        size = start;
	
        printf("------------------------------------------------------------------\n");
        printf("\tTotal(MB)=\tFree(MB)+\tinit(MB)+\tAlloc(MB)\n"); 
        printf("0\t%.1f    =\t%.1f+    \t%.1f     \t <------  initial used memory \n", total_f,free_f, used_f );
        printf("------------------------------------------------------------------\n");
	int i = 0;

	do {
		
		mem_size = sizeof(int) * size * (1024*1024) + (inc*i) * (1024*1024) ; 
		hipMalloc(&devPtr, mem_size ); // MB
                hipMemGetInfo(&free,&total);
	        free_f  = float(free)   / (1024*1024) ;
        	total_f = float(total)  / (1024*1024) ;
	        used_f =  total_f-free_f ;
		used_now_f = (float)mem_size/(1024*1024)   ; 
        printf("%d\t%.1f    =\t%.1f+    \t%.1f+    \t%.1f \n", i, total_f, free_f, used_start_f, used_now_f);
		if(devPtr == NULL) {
			printf("couldn't allocate %.1f MB ", used_now_f);
			printf("Err : %s\n", hipGetErrorString(hipGetLastError()) );
			return 0;	
		} else {
			//printf("Allocated %d int's.\n", int(size));
		}
		hipFree(devPtr);
		size = (size* sizeof(int) + inc )/sizeof(int) ;  
		mem_size = sizeof(int) * size  ;
		
		i=i+1;
	} while(1);

}
