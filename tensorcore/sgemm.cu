// build with command  nvcc -lcublas -lcudart -lcurand -arch=sm_70 gemm.cu
// use max clock 

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>

int main(int argc, char* argv[]) {

int size_M, size_N, size_K ;

if(argc<4){
	printf(" plz use matrix size M, N K with command  ./sgemm  8192 8192 8192  \n");
	printf(" will run default value M=N=K=8192\n\n\n");
	size_M=8192;
	size_K=8192;
	size_N=8192;

	} else{

	size_M =  atoi(argv[1]) ;
	size_K =  atoi(argv[2]) ;
	size_N =  atoi(argv[3]) ;
}


#define SIZE size  //  4096 8192 10240 16384 24576
#define MATRIX_M size_M
#define MATRIX_N size_N
#define MATRIX_K size_K

#define num_clock 1530 // V100 16GB SMX
#define num_SM 80
#define num_CUDA 64
#define num_FMA 2
#define num_DP_ratio 2
#define FP32_OP  num_clock*num_SM*num_CUDA*num_FMA
#define FP64_OP  FP32_OP / num_DP_ratio
#define TOTAL_OP  MATRIX_M * MATRIX_N * MATRIX_K * 2
#define TOTAL_OP2 (MATRIX_M*MATRIX_N) * (2*MATRIX_K+2) 


   printf("FP32 Matrix Memory Size A %dx%d : %.1f MB   \n", MATRIX_M, MATRIX_K,   (float) (sizeof(float) * (float) (MATRIX_M*MATRIX_K)  / ( 1024 * 1024 ) )  );
   printf("FP32 Matrix Memory Size B %dx%d : %.1f MB   \n", MATRIX_K, MATRIX_N,    (float) (sizeof(float) * (float) (MATRIX_K*MATRIX_N)  / ( 1024 * 1024 ) )  );
   printf("FP32 Matrix Memory Size C %dx%d : %.1f MB   \n", MATRIX_M, MATRIX_K,    (float) (sizeof(float) * (float) (MATRIX_M*MATRIX_N)  / ( 1024 * 1024 ) )  );

	
   float *a_fp32;
   float *b_fp32;
   float *c_fp32;

   printf(" Step1. Initialize GPU API handles...\n");
   hiprandGenerator_t gen;

   hipblasHandle_t cublasHandle;
   hipblasCreate(&cublasHandle);
       
   hipEvent_t startcublas_fp32;
   hipEvent_t stopcublas_fp32;      

   hipEventCreate(&startcublas_fp32);
   hipEventCreate(&stopcublas_fp32);
   
   // Use tensor cores
   // cublasSetMathMode(cublasHandle, CUBLAS_TENSOR_OP_MATH);

   printf(" Step2. Memory Mallocation ...\n");
	 
   hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float));
   hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float));
   hipMalloc((void**)&c_fp32, MATRIX_M * MATRIX_N * sizeof(float));
	 


   printf(" Step3. Data init with cuRAND ...\n");
   hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
   hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL);

   hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K);
   hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N);
   hiprandGenerateUniform(gen, c_fp32, MATRIX_M * MATRIX_N);
	 
   hiprandDestroyGenerator(gen);

   float alpha_fp32 = 2.0f;
   float  beta_fp32 = 2.0f;

   printf(" Step5. Ready to Run...\n");
   printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha_fp32, beta_fp32);

   // Now using cuBLAS
   printf(" Step6. warm up...\n");
   hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
               MATRIX_M, MATRIX_N, MATRIX_K, 
  	       &alpha_fp32, 
	       a_fp32, MATRIX_M, 
	       b_fp32, MATRIX_K, 
	       &beta_fp32, 
	       c_fp32, MATRIX_M);
								
    printf(" Step7.  Running with cuBLAS... sgemm\n");
    hipEventRecord(startcublas_fp32);
    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
	        &alpha_fp32, 
		a_fp32, MATRIX_M, 
		b_fp32, MATRIX_K, 
		&beta_fp32, 
		c_fp32, MATRIX_M);
   hipEventRecord(stopcublas_fp32);							
   hipEventSynchronize(stopcublas_fp32);

	 
	 
   float cublasTime_fp32 ;
   hipEventElapsedTime(&cublasTime_fp32, startcublas_fp32, stopcublas_fp32);

   printf("\n\nSGEMM cublas took %fms ", cublasTime_fp32);
   printf(" with   %f OP clock %d Mhz \n", (double) TOTAL_OP,  num_clock );

  double RPEAK = (double) FP32_OP /(1000000) ;
  double RMAX = (double) ( ((double)TOTAL_OP / (double) (1000000) ) / ((double) cublasTime_fp32)/1000 );

   printf("FP32 RPeak: %.2f TFLOPS",  RPEAK );  
   printf(" SGEMM : %.2f TFLOPS\n",   RMAX  ); 

   printf("Ratio of Real/Theoretic %.2f   \n",  RMAX/RPEAK );



   hipEventDestroy(startcublas_fp32);
   hipEventDestroy(stopcublas_fp32);
   
	 
   hipFree(a_fp32);
   hipFree(b_fp32);
   hipFree(c_fp32);

   hipDeviceReset();
   return 0;
}
