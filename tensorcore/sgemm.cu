#include "hip/hip_runtime.h"
// build with command  nvcc -lcublas -lcudart -lcurand -arch=sm_70 gemm.cu
// use max clock 

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

// Must be multiples of 16 to fit TensorCore
#define SIZE 8192  //  4096 8192 10240 16384 24576
#define MATRIX_M SIZE 
#define MATRIX_N SIZE
#define MATRIX_K SIZE

#define num_clock 1530 // V100 16GB SMX
#define num_SM 80
#define num_CUDA 64
#define num_FMA 2
#define num_DP_ratio 2
#define FP32_OP  num_clock*num_SM*num_CUDA*num_FMA
#define FP64_OP  FP32_OP / num_DP_ratio
#define TOTAL_OP  MATRIX_M * MATRIX_N * MATRIX_K * 2
#define TOTAL_OP2 (MATRIX_M*MATRIX_N) * (2*MATRIX_K+2) 

int main(int argc, char* argv[]) {
   printf("FP32 Matrix Memory Size : %.1f \n",  (float) (sizeof(float) * (float) (MATRIX_M*MATRIX_M)  / ( 1024 * 1024 ) )  );
	

   float *a_fp32;
   float *b_fp32;
   float *c_fp32;

   float *c_cublas_fp32;

   float *c_host_cublas_fp32; // for error tolorence 

   printf(" Step1. Initialize GPU API handles...\n");
   hiprandGenerator_t gen;

   hipblasHandle_t cublasHandle;
   hipblasCreate(&cublasHandle);
       

   hipEvent_t startcublas_fp32;
 
   hipEvent_t stopcublas_fp32;      


   hipEventCreate(&startcublas_fp32);

   hipEventCreate(&stopcublas_fp32);
   
   // Use tensor cores
   // hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);

   printf(" Step2. Memory Mallocation ...\n");
	 
   hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float));
   hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float));
   hipMalloc((void**)&c_fp32, MATRIX_M * MATRIX_N * sizeof(float));
	 

   hipMalloc((void**)&c_cublas_fp32, MATRIX_M * MATRIX_N * sizeof(float));


   c_host_cublas_fp32 = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

   printf(" Step3. Data init with cuRAND ...\n");
   hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
   hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL);


   hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K);
   hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N);
   hiprandGenerateUniform(gen, c_fp32, MATRIX_M * MATRIX_N);

   hipMemcpy(c_cublas_fp32, c_fp32, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice);
	 
   hiprandDestroyGenerator(gen);

   float alpha_fp32 = 2.0f;
   float  beta_fp32 = 2.0f;

   printf(" Step5. Ready to Run...\n");
   printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha_fp64, beta_fp64);

   // Now using cuBLAS
   printf("warm up...");
   hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
               MATRIX_M, MATRIX_N, MATRIX_K, 
  	       &alpha_fp32, 
	       a_fp32, MATRIX_M, 
	       b_fp32, MATRIX_K, 
	       &beta_fp32, 
	       c_fp32, MATRIX_M);
								
    printf(" Step6.  Running with cuBLAS... sgemm\n");
    hipEventRecord(startcublas_fp32);
    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
	        &alpha_fp32, 
		a_fp32, MATRIX_M, 
		b_fp32, MATRIX_K, 
		&beta_fp32, 
		c_fp32, MATRIX_M);
   hipEventRecord(stopcublas_fp32);							
   hipEventSynchronize(stopcublas_fp32);

	 
   printf(" Step7. Download results...\n");

   hipMemcpy( c_host_cublas_fp32, c_cublas_fp32, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost);
	 
   float cublasTime_fp32;
   hipEventElapsedTime(&cublasTime_fp32, startcublas_fp32, stopcublas_fp32);


   printf("SGEMM cublas took %fms\n", cublasTime_fp32);

   printf(" with   %f Operation and clock %d Mhz \n", (double) TOTAL_OP,  num_clock );

   printf("FP32 RPeak: %.2f TFLOPS",  (double) FP32_OP /(1000000)   );  
   printf(" SGEMM : %.2f TFLOPS\n",   (double) ( ((double)TOTAL_OP / (double) (1000000) ) / ((double) cublasTime_fp32)/1000 ) ); 

   hipEventDestroy(startcublas_fp32);       
   hipEventDestroy(stopcublas_fp32);
	 
   hipFree(a_fp32);
   hipFree(b_fp32);
   hipFree(c_fp32);

   hipFree(c_cublas_fp32);
   free(c_host_cublas_fp32);

   hipDeviceReset();
   return 0;
}
